#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
// #include <hiprand.h>
// #include <hiprand/hiprand_kernel.h>
#include "rt_utils.cu"
#include <math.h>
#include <stdbool.h>
#include <stdio.h>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

#define VP_W 4.0f
#define VP_H VP_W * 9 / 16
#define DIAFRAGM 0.01f
#define FOCAL 6.0f
#define R_COUNT 5

__device__ void pixel_ray(double x, double y, Vec3 *origin, Vec3 *direction) {
  origin->x = 0;
  origin->y = 4.0f;
  origin->z = 0;

  direction->x = x;
  direction->y = y;
  direction->z = FOCAL;
  divide_v(direction, len_v(direction));
  rotateDirection(direction, 7, 0, 0);
  normalize_v(direction);
}

__device__ void trace_ray(Vec3 *origin, Vec3 *direction, int ray_count,
                          Object *objects, int object_count, Vec3 *ray_energy,
                          unsigned *seed) {
  Vec3 ray_color = {.x = 1, .y = 1, .z = 1};

  Vec3 intersection, normal;
  int hit_index, reflect_count, prev_hit_index = -1;
  Vec3 r_o, r_d, emitted_light;

  Vec3 sky_color, sky_emitted_light;
  float sky_emitted_light_strength = 0.15;

  for (int i = 0; i < ray_count; i++) {
    ray_color = {.x = 1, .y = 1, .z = 1};
    reflect_count = 0;
    copy_v(&r_o, origin);
    copy_v(&r_d, direction);
    prev_hit_index = -1;
    r_o.x += my_drand(seed) * DIAFRAGM - DIAFRAGM / 2;
    r_o.y += my_drand(seed) * DIAFRAGM - DIAFRAGM / 2;
    normalize_v(&r_d);
    while (reflect_count < 15) {
      if (find_closest_hit(&r_o, &r_d, objects, object_count, prev_hit_index,
                           &intersection, &normal, &hit_index)) {
        reflect_count++;
        Object *obj = &objects[hit_index];
        prev_hit_index = hit_index;

        copy_v(&r_o, &intersection);
        reflect(&r_d, &normal, &r_d);
        random_direction_hemi_and_lerp(&r_d, &normal, seed,
                                       1.0 - obj->material.specular_rate);
        normalize_v(&r_d);

        emitted_light.x = obj->material.emission_color.a;
        emitted_light.y = obj->material.emission_color.b;
        emitted_light.z = obj->material.emission_color.c;
        mult_v(&emitted_light, obj->material.emission_strength);

        mult_v(&emitted_light, &ray_color);
        add_v(ray_energy, &emitted_light);
        float max_e = max(ray_energy->x, max(ray_energy->y, ray_energy->z));

        ray_color.x *= obj->material.color.a;
        ray_color.y *= obj->material.color.b;
        ray_color.z *= obj->material.color.c;
        float max_c = max(ray_color.x, max(ray_color.y, ray_color.z));
        if (max_c > 1) {
          ray_color.x /= max_c;
          ray_color.y /= max_c;
          ray_color.z /= max_c;
        }
      } else {
        sky_color.x = 0.863f;
        sky_color.y = 0.949f;
        sky_color.z = 0.961f;
        sky_emitted_light.x = sky_emitted_light_strength;
        sky_emitted_light.y = sky_emitted_light_strength;
        sky_emitted_light.z = sky_emitted_light_strength;

        mult_v(&sky_emitted_light, &sky_color);

        mult_v(&sky_emitted_light, &ray_color);
        add_v(ray_energy, &sky_emitted_light);
        float max_e = max(ray_energy->x, max(ray_energy->y, ray_energy->z));

        break;
      }
    }
  }

  ray_energy->x /= ray_count;
  ray_energy->y /= ray_count;
  ray_energy->z /= ray_count;
  float max_e = max(ray_energy->x, max(ray_energy->y, ray_energy->z));
  if (max_e > 1) {
    ray_energy->x /= max_e;
    ray_energy->y /= max_e;
    ray_energy->z /= max_e;
  }
}

__global__ void test_kernel(Object *objects, int count, UCHAR *r, UCHAR *g,
                            UCHAR *b, int w, int h, int rays) {
  int x_p = (blockDim.x * blockIdx.x + threadIdx.x) / rays;
  int y_p = blockDim.y * blockIdx.y + threadIdx.y;

  if (x_p >= w || y_p >= h)
    return;
  int index = blockDim.x * blockIdx.x + threadIdx.x + y_p * (w * rays);
  unsigned int seed = index + 10;

  double x = ((x_p - w / 2.0) / w) * VP_W * 2,
         y = -((y_p - h / 2.0) / h) * VP_H * 2;

  Vec3 r_origin;
  Vec3 r_dir;
  pixel_ray(x, y, &r_origin, &r_dir);

  Vec3 ray_energy = {.x = 0, .y = 0, .z = 0};
  trace_ray(&r_origin, &r_dir, R_COUNT, objects, count, &ray_energy, &seed);

  r[index] = (ray_energy.x > 1 ? 1 : ray_energy.x) * 255.0;
  g[index] = (ray_energy.y > 1 ? 1 : ray_energy.y) * 255.0;
  b[index] = (ray_energy.z > 1 ? 1 : ray_energy.z) * 255.0;
}

__global__ void average_kernel(UCHAR *r, UCHAR *g, UCHAR *b, UCHAR *r_out,
                               UCHAR *g_out, UCHAR *b_out, int w, int h,
                               int rays) {
  int x_p = blockDim.x * blockIdx.x + threadIdx.x;
  int y_p = blockDim.y * blockIdx.y + threadIdx.y;
  if (x_p >= w || y_p >= h)
    return;
  int index = x_p + y_p * w;

  int rp = 0;
  int gp = 0;
  int bp = 0;
  for (int i = 0; i < rays; i++) {
    int in_index = (x_p * rays) + i + y_p * (w * rays);
    rp += r[in_index];
    gp += g[in_index];
    bp += b[in_index];
  }

  r_out[index] = rp / rays;
  g_out[index] = gp / rays;
  b_out[index] = bp / rays;
}

void test_renderer(Scene *scene, Frame *frame, PipelineSetting setting) {
  int w = frame->width;
  int h = frame->height;
  int rays = 5;
  UCHAR *r;
  hipMalloc(&r, sizeof(UCHAR) * w * h * rays);
  UCHAR *g;
  hipMalloc(&g, sizeof(UCHAR) * w * h * rays);
  UCHAR *b;
  hipMalloc(&b, sizeof(UCHAR) * w * h * rays);

  UCHAR *r_out;
  hipMalloc(&r_out, sizeof(UCHAR) * w * h);
  UCHAR *g_out;
  hipMalloc(&g_out, sizeof(UCHAR) * w * h);
  UCHAR *b_out;
  hipMalloc(&b_out, sizeof(UCHAR) * w * h);

  Object *d_objects;
  hipMalloc(&d_objects, sizeof(Object) * scene->count);
  hipMemcpy(d_objects, scene->objects, sizeof(Object) * scene->count,
             hipMemcpyHostToDevice);

  int block_size = 16;
  dim3 thd = dim3(block_size, block_size);
  dim3 bld = dim3((w * rays - 1) / block_size + 1, (h - 1) / block_size + 1);
  printf("%d, %d\n", w, h);
  printf("%d, %d\n", bld.x, bld.y);

  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  test_kernel<<<bld, thd>>>(d_objects, scene->count, r, g, b, w, h, rays);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("GPU kernel took %.4f ms \n\n", time);

  // ----
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  average_kernel<<<bld, thd>>>(r, g, b, r_out, g_out, b_out, w, h, rays);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("Average kernel took %.4f ms \n\n", time);

  hipMemcpy(frame->r, r_out, w * h, hipMemcpyDeviceToHost);
  hipMemcpy(frame->g, g_out, w * h, hipMemcpyDeviceToHost);
  hipMemcpy(frame->b, b_out, w * h, hipMemcpyDeviceToHost);
  hipFree(r);
  hipFree(g);
  hipFree(b);
}

int main() {
  int width = 1200;
  int height = width * 9 / 16;
  PipelineSetting setting = {.width = width,
                             .height = height,
                             .debug = 1,
                             .save = 1,
                             .out_file = (char *)"test_cu.bmp"};
  Scene *scene = sample_scene_cuda();

  pipeline(scene, setting, test_renderer);

  free_scene(scene);
}
